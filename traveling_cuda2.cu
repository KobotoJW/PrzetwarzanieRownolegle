#include <iostream>
#include <vector>
#include <chrono>
#include <omp.h>
#include <limits.h>
#include <hip/hip_runtime.h>

__global__ void tspKernel(int *d_costMatrix, int *d_results, int *d_paths, int numCities) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < numCities) {
        int* path = new int[numCities];
        int cost = 0;

        path[0] = tid; // Start from the city 'tid'
        for (int i = 1; i < numCities; i++) {
            int minCost = INT_MAX;
            int nextCity = -1;
            for (int j = 0; j < numCities; j++) {
                bool visited = false;
                for (int k = 0; k < i; k++) {
                    if (path[k] == j) {
                        visited = true;
                        break;
                    }
                }
                if (!visited && j != path[i-1]) {
                    int newCost = d_costMatrix[path[i-1] * numCities + j];
                    if (newCost < minCost) {
                        minCost = newCost;
                        nextCity = j;
                    }
                }
            }
            path[i] = nextCity;
            cost += d_costMatrix[path[i-1] * numCities + path[i]];
        }

        d_results[tid] = cost;

        for (int i = 0; i < numCities; i++) {
            d_paths[tid * numCities + i] = path[i];
        }

        delete[] path;
    }
}

std::vector<std::vector<int>> gen_matrix(int num_cities) {
    srand(time(0)); // Seed

    std::vector<std::vector<int>> matrix(num_cities, std::vector<int>(num_cities));

    #pragma omp parallel for
    for (int i = 0; i < num_cities; ++i) {
        unsigned int seed = time(0) ^ (i + omp_get_thread_num());
        for (int j = i + 1; j < num_cities; ++j) {
            if (i == j) {
                matrix[i][j] = 0;
            } else {
                int cost = rand_r(&seed) % 100 + 1;

                matrix[i][j] = cost;
                matrix[j][i] = cost;
            }
        }
    }

    // //Print the matrix
    // for (int i = 0; i < num_cities; ++i) {
    //     for (int j = 0; j < num_cities; ++j) {
    //         std::cout << matrix[i][j] << ' ';
    //     }
    //     std::cout << '\n';
    // }

    return matrix;
}

int main(int argc, char **argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <num_cities>\n";
        return 1;
    }

    int numCities;
    try {
        numCities = std::stoi(argv[1]);
    } catch (std::invalid_argument &e) {
        std::cerr << "Invalid number of cities\n";
        return 1;
    }

    int MAX_THREADS = numCities;
    int BLOCK_SIZE = 256;

    int* h_costMatrix = new int[numCities * numCities];
    int *d_costMatrix;

    std::vector<std::vector<int>> matrix = gen_matrix(numCities);
    for (int i = 0; i < numCities; i++) {
        for (int j = 0; j < numCities; j++) {
            h_costMatrix[i * numCities + j] = matrix[i][j];
        }
    }

    hipMalloc(&d_costMatrix, sizeof(int) * numCities * numCities);
    hipMemcpy(d_costMatrix, h_costMatrix, sizeof(int) * numCities * numCities, hipMemcpyHostToDevice);

    int* h_results = new int[MAX_THREADS];
    int *d_results;
    hipMalloc(&d_results, sizeof(int) * MAX_THREADS);

    int *d_paths;
    hipMalloc(&d_paths, sizeof(int) * numCities * MAX_THREADS);

    int numBlocks = (numCities + BLOCK_SIZE - 1) / BLOCK_SIZE;

    auto start = std::chrono::high_resolution_clock::now();
    tspKernel<<<numBlocks, BLOCK_SIZE>>>(d_costMatrix, d_results, d_paths, numCities);
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Elapsed time: " << elapsed.count() << " µs\n";

    hipMemcpy(h_results, d_results, sizeof(int) * MAX_THREADS, hipMemcpyDeviceToHost);

    int* h_paths = new int[MAX_THREADS * numCities];
    hipMemcpy(h_paths, d_paths, sizeof(int) * numCities * MAX_THREADS, hipMemcpyDeviceToHost);

    int minCost = INT_MAX;
    int minCostIndex = 0;
    for (int i = 0; i < numCities; i++) {
        if (h_results[i] < minCost) {
            minCost = h_results[i];
            minCostIndex = i;
        }
    }

    // // Print the cheapest path
    // printf("Cheapest path: ");
    // for (int i = 0; i < numCities; i++) {
    //     printf("%d ", h_paths[minCostIndex * numCities + i]);
    // }
    // printf("\n");

    delete[] h_costMatrix;
    delete[] h_results;
    delete[] h_paths;

    hipFree(d_paths);

    hipFree(d_costMatrix);
    hipFree(d_results);

    printf("Minimum cost: %d\n", minCost);
    return 0;
}
